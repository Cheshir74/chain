#include "hip/hip_runtime.h"
#include "BytomPoW.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"

void print_floatMatrix(const char *s, float *devPtr) {

  float f[256 * 256];
  hipblasGetMatrix (256, 256, sizeof(float), devPtr, 256, f, 256);
  printf("%s:\n", s);
  for(int i =0; i<256; i++) {
    if(0 == (i%8)) printf("\n");
    printf("%3.1f\t", f[i]);
  }
  printf("\n");
}

void print_int8HostMatrix(const char *s, int8_t *i8Ptr) {

  printf("%s:\n", s);
  for(int i =0; i<256; i++) {
    if(0 == (i%8)) printf("\n");
    printf("%d\t", i8Ptr[i]);
  }
  printf("\n");
}

void initMatVecGpu(BytomMatListGpu* matListGpu_float, BytomMatList* matList_int8) {

  float *tmp = (float *)malloc(sizeof(float) * 256 * 256);
  for(int i=0; i<matList_int8->matVec.size(); i++) {
    int8_t* hostPtr_i8 = (int8_t*)(matList_int8->at(i).d);
    for(int k = 0; k < 256 * 256; k++)
      tmp[k] = hostPtr_i8[k];

    float* devPtr = (float*)(matListGpu_float->at(i));
    hipblasStatus_t stat = hipblasSetMatrix (256, 256, sizeof(*devPtr), tmp, 256, devPtr, 256);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      std::cerr<<"Fail to Set CuBlas Matrix."<<std::endl;
      exit(EXIT_FAILURE);
    }
  }
}

__global__ void converInt32ToInt8_gpu(float * in, float* out) {
  int32_t data_i32 = in[blockIdx.x * blockDim.x + threadIdx.x];
  int8_t data_i8 = ((data_i32&0xFF)+ ((data_i32>>8)&0xFF))&0xFF;
  out[blockIdx.x * blockDim.x + threadIdx.x] = data_i8;
}

void core_mineBytom_gpu(
    std::vector<uint8_t> fourSeq[4],
    BytomMatListGpu* matListGpu_float,
    uint32_t data[64],
    hipblasHandle_t handle) {

  Mat256x256float *idt=new Mat256x256float;
  Mat256x256float *mat=new Mat256x256float;
  Mat256x256float *tmp=new Mat256x256float;
  Mat256x256float *res=new Mat256x256float[4];
  idt->toIdentityMatrix();

  float* devIdt;
  float* devTmp;
  float* devTmp_r;
  hipMalloc ((void**)&devIdt, 256*256*sizeof(*devIdt));
  hipMalloc ((void**)&devTmp, 256*256*sizeof(*devTmp));
  hipMalloc ((void**)&devTmp_r, 256*256*sizeof(*devTmp_r));
  hipblasStatus_t stat = hipblasSetMatrix (256, 256, sizeof(float), idt->f, 256, devIdt, 256); //HKKUO: A). Memory Set
  const float alpha = 1;
  const float beta = 0;

  for(int k=0; k<4; k++) {
    for(int j=0; j<2; j++) {
      for(int i=0; i<32; i+=2) {
        if (j==0 && i==0)
          stat = hipblasSgemm(handle,
                              HIPBLAS_OP_N,
                              HIPBLAS_OP_N,
                              256,
                              256,
                              256,
                              &alpha,
                              matListGpu_float->at(fourSeq[k][i]),
                              256,
                              devIdt,
                              256,
                              &beta,
                              devTmp_r,
                              256);  //HKKUO: B). General Matrix Multiplication (GEMM)
        else
          stat = hipblasSgemm(handle,
                              HIPBLAS_OP_N,
                              HIPBLAS_OP_N,
                              256,
                              256,
                              256,
                              &alpha,
                              matListGpu_float->at(fourSeq[k][i]),
                              256,
                              devTmp,
                              256,
                              &beta,
                              devTmp_r,
                              256);  //HKKUO: B). General Matrix Multiplication (GEMM)
        if (stat != HIPBLAS_STATUS_SUCCESS) {
          std::cerr<<"Fail to Run CuBlas GemmEx1."<<std::endl;
          exit(EXIT_FAILURE);
        }
        converInt32ToInt8_gpu<<<256, 256>>>(devTmp_r, devTmp);
        stat = hipblasSgemm(handle,
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            256,
                            256,
                            256,
                            &alpha,
                            matListGpu_float->at(fourSeq[k][i+1]),
                            256,
                            devTmp,
                            256,
                            &beta,
                            devTmp_r,
                            256);  //HKKUO: B). General Matrix Multiplication (GEMM)
        if (stat != HIPBLAS_STATUS_SUCCESS) {
          std::cerr<<"Fail to Run CuBlas GemmEx2."<<std::endl;
          exit(EXIT_FAILURE);
        }
        converInt32ToInt8_gpu<<<256, 256>>>(devTmp_r, devTmp);
      }
    }
    stat = hipblasGetMatrix (256, 256, sizeof(*devTmp), devTmp, 256, res[k].f, 256);
  }

  mat->add(res[0], res[1]);  //HKKUO: C). Matrix Addition
  tmp->add(*mat, res[2]);    //HKKUO: C). Matrix Addition
  mat->add(*tmp, res[3]);    //HKKUO: C). Matrix Addition

  Arr256x64i32 arr(*mat);
  arr.reduceFNV();           //HKKUO: D). Reduction
  arr.fillWithD0(data);      //HKKUO: E). Memory Set
  delete mat;
  delete tmp;
  delete[] res;
}
