#include "hip/hip_runtime.h"
#include "BytomPoW.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"

void print_floatMatrix(const char *s, float *devPtr) {

  float f[256 * 256];
  hipblasGetMatrix (256, 256, sizeof(float), devPtr, 256, f, 256);
  printf("%s:\n", s);
  for(int i =0; i<256; i++) {
    if(0 == (i%8)) printf("\n");
    printf("%3.1f\t", f[i]);
  }
  printf("\n");
}

void print_int8HostMatrix(const char *s, int8_t *i8Ptr) {

  printf("%s:\n", s);
  for(int i =0; i<256; i++) {
    if(0 == (i%8)) printf("\n");
    printf("%d\t", i8Ptr[i]);
  }
  printf("\n");
}

void initMatVecGpu(BytomMatListGpu* matListGpu_float, BytomMatList* matList_int8) {

  float *tmp = (float *)malloc(sizeof(float) * 256 * 256);
  for(int i=0; i<matList_int8->matVec.size(); i++) {
    int8_t* hostPtr_i8 = (int8_t*)(matList_int8->at(i).d);
    for(int k = 0; k < 256 * 256; k++)
      tmp[k] = hostPtr_i8[k];

    float* devPtr = (float*)(matListGpu_float->at(i));
    hipblasStatus_t stat = hipblasSetMatrix (256, 256, sizeof(*devPtr), tmp, 256, devPtr, 256);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      std::cerr<<"Fail to Set CuBlas Matrix."<<std::endl;
      exit(EXIT_FAILURE);
    }
  }
}

__global__ void converInt32ToInt8_gpu(float * in, float* out) {
  int32_t data_i32 = in[blockIdx.x * blockDim.x + threadIdx.x];
  int8_t data_i8 = ((data_i32&0xFF)+ ((data_i32>>8)&0xFF))&0xFF;
  out[blockIdx.x * blockDim.x + threadIdx.x] = data_i8;
}

void core_mineBytom_gpu(
    std::vector<uint8_t> fourSeq[4],
    BytomMatListGpu* matListGpu_float,
    uint32_t data[64],
    hipblasHandle_t handle) {

  Mat256x256float *idt=new Mat256x256float;
  Mat256x256float *mat=new Mat256x256float;
  Mat256x256float *tmp=new Mat256x256float;
  Mat256x256float *res=new Mat256x256float[4];
  idt->toIdentityMatrix();

  float* devIdt;
  float* devTmp;
  float* devTmp_r;
  float* devIn;
  hipMalloc ((void**)&devIn, 4*256*256*sizeof(*devIn));
  hipMalloc ((void**)&devIdt, 4*256*256*sizeof(*devIdt));
  hipMalloc ((void**)&devTmp, 4*256*256*sizeof(*devTmp));
  hipMalloc ((void**)&devTmp_r, 4*256*256*sizeof(*devTmp_r));
  hipblasStatus_t stat = hipblasSetMatrix (256, 256, sizeof(float), idt->f, 256, devIdt, 256); //HKKUO: A). Memory Set
  stat = hipblasSetMatrix (256, 256, sizeof(float), idt->f, 256, devIdt+4*256*256, 256);
  stat = hipblasSetMatrix (256, 256, sizeof(float), idt->f, 256, devIdt+8*256*256, 256);
  stat = hipblasSetMatrix (256, 256, sizeof(float), idt->f, 256, devIdt+12*256*256, 256);
  const float alpha = 1;
  const float beta = 0;

  //for(int k=0; k<4; k++) {
    for(int j=0; j<2; j++) {
      for(int i=0; i<32; i+=2) {

        for(int k=0; k<4; k++)
          hipblasScopy(handle, 256*256, matListGpu_float->at(fourSeq[k][i]), 4, devIn+i*4*256*256, 4);

        if (j==0 && i==0)
          stat = hipblasSgemm(handle,
                              HIPBLAS_OP_N,
                              HIPBLAS_OP_N,
                              256 * 1,
                              256,
                              256,
                              &alpha,
                              devIn,
                              256*1,
                              devIdt,
                              256,
                              &beta,
                              devTmp_r,
                              256*1);  //HKKUO: B). General Matrix Multiplication (GEMM)
        else
          stat = hipblasSgemm(handle,
                              HIPBLAS_OP_N,
                              HIPBLAS_OP_N,
                              256 * 1,
                              256,
                              256,
                              &alpha,
                              devIn,
                              256*1,
                              devTmp,
                              256,
                              &beta,
                              devTmp_r,
                              256*1);  //HKKUO: B). General Matrix Multiplication (GEMM)
        if (stat != HIPBLAS_STATUS_SUCCESS) {
          printf("[%d]", stat);
          std::cerr<<"Fail to Run CuBlas Gemm1."<<std::endl;
          exit(EXIT_FAILURE);
        }
        converInt32ToInt8_gpu<<<256*4, 256>>>(devTmp_r, devTmp);

        for(int k=0; k<4; k++)
          hipblasScopy(handle, 256*256, matListGpu_float->at(fourSeq[k][i]), 4, devIn+i*4*256*256, 4);

        stat = hipblasSgemm(handle,
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            256 * 4,
                            256,
                            256,
                            &alpha,
                            devIn,
                            256*4,
                            devTmp,
                            256,
                            &beta,
                            devTmp_r,
                            256*4);  //HKKUO: B). General Matrix Multiplication (GEMM)
        if (stat != HIPBLAS_STATUS_SUCCESS) {
          printf("[%d]", stat);
          std::cerr<<"Fail to Run CuBlas Gemm2."<<std::endl;
          exit(EXIT_FAILURE);
        }
        converInt32ToInt8_gpu<<<256*4, 256>>>(devTmp_r, devTmp);
      }
    }

  for(int k=0; k<4; k++)
    stat = hipblasGetMatrix (256, 256, sizeof(*devTmp), devTmp + k*4*256*256, 256, res[k].f, 256);
  //}

  mat->add(res[0], res[1]);  //HKKUO: C). Matrix Addition
  tmp->add(*mat, res[2]);    //HKKUO: C). Matrix Addition
  mat->add(*tmp, res[3]);    //HKKUO: C). Matrix Addition

  Arr256x64i32 arr(*mat);
  arr.reduceFNV();           //HKKUO: D). Reduction
  arr.fillWithD0(data);      //HKKUO: E). Memory Set
  delete mat;
  delete tmp;
  delete[] res;
}
